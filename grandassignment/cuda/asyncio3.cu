#include <cmath>
#include <cstdio>
#include <cstring>
#include <string>
#include <algorithm>
#include <iostream>
#include <cstdlib>
// #include <windows.h>
// #include <unistd.h>

#include <hip/hip_runtime.h>


// #include <device_functions.h>
#include <hip/hip_runtime_api.h>

using namespace std;

typedef double ld;
typedef long long LL;

// const int max_share_size = 512, chunk_size = 1 << 16;
const int chunk_size = 1<<16;


namespace io_impl
{
inline bool maybe_digit(char c)
{
    return c >= '0' && c <= '9';
}

struct io_s
{
private:
    FILE *fin;
    FILE *fout;

    bool negative;
    bool ok;
    char ch;

    inline char next_char()
    {
        static char buf[100000], *p1 = buf, *p2 = buf;
        return p1 == p2 && (p2 = (p1 = buf) + fread(buf, 1, 100000, fin), p1 == p2) ? EOF : *p1++;
    }

public:
    void init(FILE *_in, FILE *_out)
    {
        fin = _in;
        fout = _out;
        ch = next_char();
        ok = true;
    }

    template <typename T>
    bool run(T &_v)
    {
        _v = 0;
        while (!maybe_digit(ch) && ch != EOF)
            ch = next_char();
        if (ch == EOF)
            return ok = false;
        do
        {
            _v = (_v << 1) + (_v << 3) + ch - '0';
        } while (maybe_digit(ch = next_char()));
        return true;
    }

    template <typename T>
    bool rd(T &_v)
    {
        negative = false;
        _v = 0;
        while (!maybe_digit(ch) && ch != EOF)
        {
            negative = ch == '-';
            ch = next_char();
        }
        if (ch == EOF)
            return ok = false;
        do
        {
            _v = (_v * 10) + (ch - '0');
        } while (maybe_digit(ch = next_char()));
        static double _map[] = {1, 1e-1, 1e-2, 1e-3, 1e-4, 1e-5, 1e-6};
        if (ch == '.')
        {
            int tp = 0;
            while (maybe_digit(ch = next_char()))
            {
                _v = (_v * 10) + (ch - '0');
                ++tp;
            }
            _v *= _map[tp];
        }
        if (negative)
            _v = -_v;
        return true;
    }

    // template <typename T>
    // bool rd(T &_v)
    // {
    //     negative = false;
    //     _v = 0;
    //     while (!maybe_digit(ch) && ch != EOF)
    //     {
    //         negative = ch == '-';
    //         ch = next_char();
    //     }
    //     if (ch == EOF)
    //         return ok = false;
    //     do
    //     {
    //         _v = (_v * 10) + (ch - '0');
    //     } while (maybe_digit(ch = next_char()));
    //     static int stk[70], tp;
    //     if (ch == '.')
    //     {
    //         tp = 0;
    //         T _v2 = 0;
    //         while (maybe_digit(ch = next_char()))
    //         {
    //             stk[tp++] = ch - '0';
    //         }
    //         while (tp--)
    //         {
    //             _v2 = _v2 / 10 + stk[tp];
    //         }
    //         _v += _v2 / 10;
    //     }
    //     if (negative)
    //         _v = -_v;
    //     return true;
    // }
};

} // namespace io_impl

using namespace io_impl;

io_s kbio;

namespace output {
    const int OutputBufferSize = 1 << 20;

    char buffer[OutputBufferSize];
    char *s = buffer;
    inline void flush() {
        fwrite(buffer, 1, s-buffer, stdout);
        s = buffer;
        fflush(stdout);
    }
    inline void print(const char ch) {
        // putchar(ch); return;
        if (s-buffer>OutputBufferSize-2) flush();
        *s++ = ch;
    }
    inline void print(char *str) {
        while (*str!=0) print(char(*str++));
    }
    inline void print(int x) {
        // printf("%d", x); return;
        char buf[25] = {0}, *p = buf;
        if (x<0) print('-'), x=-x;
        if (x == 0) print('0');
        while (x) *(++p) = x%10, x/=10;
        while (p != buf) print(char(*(p--)+'0'));
    }

    inline void print(ld x) {
        // printf("%.2f", x);
        static char buf[100];
        sprintf(buf, "%.2f", x);
        print(buf);
    }
}


struct ios {
    static const int IN_LEN=1<<18|1;
    char buf[IN_LEN],*s,*t; 
    inline char read(){
        return (s==t)&&(t=(s=buf)+fread(buf,1,IN_LEN,stdin)),s==t?-1:*s++;
    }
    inline bool isEOF() {   
        return (s==t)&&(t=(s=buf)+fread(buf,1,IN_LEN,stdin)),s==t;
    }
    inline ios & operator >> (int &x){
        static char c11,boo;
        for(c11=read(),boo=0;!isdigit(c11);c11=read()){
            if(c11==-1)return *this;
            boo|=c11=='-';
        }
        for(x=0;isdigit(c11);c11=read())x=x*10+(c11^'0');
        boo&&(x=-x);
        return *this;
    }

    inline ios & operator >> (LL &x){
        static char c11,boo;
        for(c11=read(),boo=0;!isdigit(c11);c11=read()){
            if(c11==-1)return *this;
            boo|=c11=='-';
        }
        for(x=0;isdigit(c11);c11=read())x=x*10+(c11^'0');
        boo&&(x=-x);
        return *this;
    }

    inline ios &operator >> (char *s) {
        int len = 0;
        char ch;
        for (ch=read(); ch=='\n' || ch == ' '; ch=read());
        if (ch == -1) {
            s[len] = 0;
            return *this;
        }
        for (; ch!='\n' && ch != ' ' && ch != -1;ch=read())
            s[len++] = ch;
        s[len] = 0;
        return *this;
    }

   inline ios &operator>>(ld &x)
    {

        char ch;
        bool neg = false, dec = false;
        double now = 0.1;
        for (ch=read(); !isdigit(ch) && (ch!='.' && ch!='-') && ch!=-1; ch=read());

        if (ch == '-') neg = true;
        else if (ch == '.') { x = 0; dec = true; }
        else if (ch != -1) x = ch-'0';
        else return *this;
        if (!dec) {
            for (ch=read(); isdigit(ch) && ch!=-1; ch=read()) {
                x = x * 10 + ch-'0';
            }
        }

        if (ch == '.')
            for (ch=read(); isdigit(ch) && ch!=-1; ch=read()) {
                x += now * (ch - '0'); now *= 0.1;
            }
        if (neg) x = -x;
        
        return *this;
    }

    inline ios &operator>>(long double &x)
    {

        char ch;
        bool neg = false, dec = false;
        double now = 0.1;
        for (ch=read(); !isdigit(ch) && (ch!='.' && ch!='-') && ch!=-1; ch=read());

        if (ch == '-') neg = true;
        else if (ch == '.') { x = 0; dec = true; }
        else if (ch != -1) x = ch-'0';
        else return *this;
        if (!dec) {
            for (ch=read(); isdigit(ch) && ch!=-1; ch=read()) {
                x = x * 10 + ch-'0';
            }
        }

        if (ch == '.')
            for (ch=read(); isdigit(ch) && ch!=-1; ch=read()) {
                x += now * (ch - '0'); now *= 0.1;
            }
        if (neg) x = -x;
        
        return *this;
    }
} io;

inline void handleCudaError(hipError_t err, string name = "fuck") {
    if (err != hipSuccess) {
        cerr << name << endl;
        cerr << hipGetErrorString(err) << endl;
        exit(0);
    }
}

ld *d_a, *d_b, *d_c, *h_a, *h_b, *h_c;
int an, am, bn, bm;
int n, m;

void copyMatrix(ld *&src,  ld *&dst, int n, int m) {
    int size = sizeof(ld) * n * m;
    
    handleCudaError(hipMalloc(&dst, size), "hipMalloc in copyMatrix");
    handleCudaError(hipMemcpy(dst, src, size, hipMemcpyHostToDevice), "memcpy in copyMatrix");
}

void copyMatrixAsync(ld *&src, ld *&dst, int n, int m, hipStream_t &stream) {
    int size = sizeof(ld) * n * m;
    handleCudaError(hipMalloc(&dst, size), "hipMalloc in copyMatrix");
    handleCudaError(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream), "memcpyasync in copyMatrix");
}

template<typename T>
__global__ void matrixMult(T *d_a, T *d_b, T *d_c, int an, int bm, int am) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i = index / bm, j = index % bm;
    if (i >= an || j >= bm) return;
    register ld sum = 0;
    int basea = i * am;
   
    for (int k=0; k<am; ++k)
        sum += d_a[basea + k] * d_b[k * bm + j];

    d_c[i * bm + j] = sum;
    // int index = threadIdx.x;
    // if (index < an * bm)
    //     d_c[index] = 1; 
}

void outputMatrix(ld *a, int n, int m) {
    // output::print(n); output::print(',');
    // output::print(m); output::print('\n');
    for (int i=0; i<n; ++i) {
        int base = i * m;
        output::print(a[base]);
        for (int j=1; j<m; ++j) {
            output::print(',');
            output::print(a[base + j]);
        }
        output::print('\n');
    }
}


void outputinterval(ld *c, int l, int r) {
    // printf("%p %d %d, %d %d\n", c, l, r, n, m);
    // printf("%.2lf\n", c[1]);
    // exit(0);
    if (l == 0) {
        // output::print('\n');
        output::print(c[l++]);
    }
    for (register int i=l; i<r; ++i) {
        if (i % m == 0) output::print('\n');
        else output::print(',');
        output::print(c[i]);
    }
    // output::print('\n');
    // output::flush();
    // exit(0);
}
void outputMatrixAsync(ld *&a, ld *&d_a, int n, int m) {



    int st = 0, ed = n * m;
    // printf("st=%d ed=%d, a=%p\n", st, ed, a); 
    hipStream_t stream[2];
    int mask = 0;
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);
    int size;
    
    for (; st<ed; st+=size, mask^=1) {
        size = min(chunk_size, ed - st);
        // printf("st=%d st+size=%d, mask=%d\n", st, st+size, mask);
        // handleCudaError(cudaMemcpy(a + st, d_a + st, size * sizeof(ld), cudaMemcpyDeviceToHost));
        handleCudaError(hipMemcpyAsync(a + st, d_a + st, size * sizeof(ld), hipMemcpyDeviceToHost, stream[mask]));
        // exit(0);
        if (st - chunk_size >= 0) {
            printf("%d %d\n",st-chunk_size, st);
            handleCudaError(hipStreamSynchronize(stream[mask^1]));
            outputinterval(a, st-chunk_size, st);
        }
    }
    st -= size;
    // sleep(1000);
    handleCudaError(hipStreamSynchronize(stream[0]));
    handleCudaError(hipStreamSynchronize(stream[1]));
    
    outputinterval(a, st, ed);
    output::print('\n');
}

int main()
{
    // #ifndef Weaverzhu
    // freopen("input.txt", "r", stdin);
    // freopen("output.txt", "w", stdout);
    kbio.init(fopen("input.txt", "r"), fopen("output.txt", "w"));


    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cerr << prop.name << endl;

    hipStream_t s_a, s_b;
    // cudaStreamCreate(&s_a);
    // cudaStreamCreate(&s_b);
    hipStreamCreateWithFlags(&s_a, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s_b, hipStreamNonBlocking);

    // #endif
    io >> an >> am; h_a = (ld*)malloc(sizeof(ld) * an * am);
    for (int i=0; i<an; ++i)
    for (int j=0; j<am; ++j)
        io >> h_a[i*am + j];
    // copyMatrix(d_a, h_a, an, am);
    copyMatrixAsync(h_a, d_a, an, am, s_a);


    io >> bn >> bm; h_b = (ld*)malloc(sizeof(ld) * bn * bm);
    for (int i=0; i<bn; ++i)
    for (int j=0; j<bm; ++j)
        io >> h_b[i*bm + j];

    // copyMatrix(h_a, d_a, an, am);
    // copyMatrix(h_b, d_b, bn, bm);
    copyMatrixAsync(h_b, d_b, bn, bm, s_b);
    n = an;
    m = bm;
    int block_size = prop.maxThreadsPerBlock, grids = (n * m + block_size - 1) / block_size;


    hipStreamSynchronize(s_a);
    hipStreamSynchronize(s_b);
    
    handleCudaError(hipMalloc(&d_c, sizeof(ld) * n * m), "allocate for h_c");

    matrixMult<<<grids, block_size>>>(d_a, d_b, d_c, an, bm, am);
    h_c = (ld*)malloc(sizeof(ld) * n * m);
    // int size = sizeof(ld) * n * m;
    // cerr << "before outputmatrixasync" << endl;

    int size = sizeof(ld) * n * m;
    // cudaStream_t stream;
    // cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking);
    // cudaStreamCreate(&stream);
    // handleCudaError(cudaMemcpyAsync(h_c, d_c, size, cudaMemcpyDeviceToHost, stream));
    // handleCudaError(cudaStreamSynchronize(stream));
    // outputinterval(h_c, 0, n * m);

    handleCudaError(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost), "memcpy back");
    // printf("h_c=%p\n", h_c);
    // outputMatrix(h_c, n, m);
    outputMatrixAsync(h_c, d_c, n, m);
    output::flush();
    
    return 0;
}



