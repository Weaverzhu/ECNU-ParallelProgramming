#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstring>
#include <string>
#include <algorithm>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

using namespace std;

typedef double ld;
typedef long long LL;


namespace io_impl
{
inline bool maybe_digit(char c)
{
    return c >= '0' && c <= '9';
}

struct io_s
{
private:
    FILE *fin;
    FILE *fout;

    bool negative;
    bool ok;
    char ch;

    inline char next_char()
    {
        static char buf[100000], *p1 = buf, *p2 = buf;
        return p1 == p2 && (p2 = (p1 = buf) + fread(buf, 1, 100000, fin), p1 == p2) ? EOF : *p1++;
    }

public:
    void init(FILE *_in, FILE *_out)
    {
        fin = _in;
        fout = _out;
        ch = next_char();
        ok = true;
    }

    template <typename T>
    bool run(T &_v)
    {
        _v = 0;
        while (!maybe_digit(ch) && ch != EOF)
            ch = next_char();
        if (ch == EOF)
            return ok = false;
        do
        {
            _v = (_v << 1) + (_v << 3) + ch - '0';
        } while (maybe_digit(ch = next_char()));
        return true;
    }

    template <typename T>
    bool rd(T &_v)
    {
        negative = false;
        _v = 0;
        while (!maybe_digit(ch) && ch != EOF)
        {
            negative = ch == '-';
            ch = next_char();
        }
        if (ch == EOF)
            return ok = false;
        do
        {
            _v = (_v * 10) + (ch - '0');
        } while (maybe_digit(ch = next_char()));
        static double _map[] = {1, 1e-1, 1e-2, 1e-3, 1e-4, 1e-5, 1e-6};
        if (ch == '.')
        {
            int tp = 0;
            while (maybe_digit(ch = next_char()))
            {
                _v = (_v * 10) + (ch - '0');
                ++tp;
            }
            _v *= _map[tp];
        }
        if (negative)
            _v = -_v;
        return true;
    }

    // template <typename T>
    // bool rd(T &_v)
    // {
    //     negative = false;
    //     _v = 0;
    //     while (!maybe_digit(ch) && ch != EOF)
    //     {
    //         negative = ch == '-';
    //         ch = next_char();
    //     }
    //     if (ch == EOF)
    //         return ok = false;
    //     do
    //     {
    //         _v = (_v * 10) + (ch - '0');
    //     } while (maybe_digit(ch = next_char()));
    //     static int stk[70], tp;
    //     if (ch == '.')
    //     {
    //         tp = 0;
    //         T _v2 = 0;
    //         while (maybe_digit(ch = next_char()))
    //         {
    //             stk[tp++] = ch - '0';
    //         }
    //         while (tp--)
    //         {
    //             _v2 = _v2 / 10 + stk[tp];
    //         }
    //         _v += _v2 / 10;
    //     }
    //     if (negative)
    //         _v = -_v;
    //     return true;
    // }
};

} // namespace io_impl

using namespace io_impl;

io_s iokb;

namespace output {
    const int OutputBufferSize = 1e6+5;

    char buffer[OutputBufferSize];
    char *s = buffer;
    inline void flush() {
        fwrite(buffer, 1, s-buffer, stdout);
        s = buffer;
        fflush(stdout);
    }
    inline void print(const char ch) {
        // putchar(ch); return;
        if (s-buffer>OutputBufferSize-2) flush();
        *s++ = ch;
    }
    inline void print(char *str) {
        while (*str!=0) print(char(*str++));
    }
    inline void print(int x) {
        // printf("%d", x); return;
        char buf[25] = {0}, *p = buf;
        if (x<0) print('-'), x=-x;
        if (x == 0) print('0');
        while (x) *(++p) = x%10, x/=10;
        while (p != buf) print(char(*(p--)+'0'));
    }

    inline void print(ld x) {
        // printf("%.2f", x);
        static char buf[100];
        sprintf(buf, "%.2f", x);
        print(buf);
    }
}


struct ios {
    
    inline ios & operator >> (int &x){
        iokb.run(&x);
        return *this;
    }


   inline ios &operator>>(ld &x)
    {
        iokb.rd(x);
        
        return *this;
    }
} io;

inline void handleCudaError(hipError_t err, string name = "fuck") {
    if (err != hipSuccess) {
        cerr << name << endl;
        cerr << hipGetErrorString(err) << endl;
        exit(0);
    }
}

ld *d_a, *d_b, *d_c, *h_a, *h_b, *h_c;
int an, am, bn, bm;
int n, m;

void copyMatrix(ld *&src,  ld *&dst, int n, int m) {
    int size = sizeof(ld) * n * m;
    
    handleCudaError(hipMalloc(&dst, size), "hipMalloc in copyMatrix");
    handleCudaError(hipMemcpy(dst, src, size, hipMemcpyHostToDevice), "memcpy in copyMatrix");

}

template<typename T>
__global__ void matrixMult(T *d_a, T *d_b, T *d_c, int an, int bm, int am) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i = index / bm, j = index % bm;
    if (i >= an || j >= bm) return;
    ld sum = 0;
    if (i < an && j < bm) {
        for (int k=0; k<am; ++k)
            sum += d_a[i * am + k] * d_b[k * bm + j];
    }
    if (i * bm + j < an * bm)
        d_c[i * bm + j] = sum;
    // int index = threadIdx.x;
    // if (index < an * bm)
    //     d_c[index] = 1; 
}

void outputMatrix(ld *a, int n, int m) {
    // output::print(n); output::print(',');
    // output::print(m); output::print('\n');
    for (int i=0; i<n; ++i) {
        int base = i * m;
        output::print(a[base]);
        for (int j=1; j<m; ++j) {
            output::print(',');
            output::print(a[base + j]);
        }
        output::print('\n');
    }
}

int main()
{
    // #ifndef Weaverzhu
    freopen("input.txt", "r", stdin);
    freopen("output.txt", "w", stdout);

    iokb.init(fopen("input.txt", "r"), fopen("output.txt", "w"));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cerr << prop.name << endl;
    // #endif
    io >> an >> am; h_a = (ld*)malloc(sizeof(ld) * an * am);
    for (int i=0; i<an; ++i)
    for (int j=0; j<am; ++j)
        io >> h_a[i*am + j];

    io >> bn >> bm; h_b = (ld*)malloc(sizeof(ld) * bn * bm);
    for (int i=0; i<bn; ++i)
    for (int j=0; j<bm; ++j)
        io >> h_b[i*bm + j];
    // B.readtrans();

    // outputMatrix(h_a, an, am);
    // outputMatrix(h_b, bn, bm);

    
    n = an;
    m = bm;
    int block_size = prop.maxThreadsPerBlock, grids = (n * m + block_size - 1) / block_size;
    copyMatrix(h_a, d_a, an, am);
    copyMatrix(h_b, d_b, bn, bm);
    handleCudaError(hipMalloc(&d_c, sizeof(ld) * n * m), "allocate for h_c");

    matrixMult<<<grids, block_size>>>(d_a, d_b, d_c, an, bm, am);
    h_c = (ld*)malloc(sizeof(ld) * n * m);
    int size = sizeof(ld) * n * m;


    handleCudaError(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost), "memcpy back");
    
    outputMatrix(h_c, n, m);
    output::flush();
    
    return 0;
}



