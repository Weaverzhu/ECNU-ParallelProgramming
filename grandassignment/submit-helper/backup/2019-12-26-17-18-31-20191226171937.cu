#include <cmath>
#include <cstdio>
#include <cstring>
#include <string>
#include <algorithm>
#include <iostream>
#include <cstdlib>

// #include <unistd.h>
// #include <windows.h>
// #include <unistd.h>

#include <hip/hip_runtime.h>


// #include <device_functions.h>
#include <hip/hip_runtime_api.h>

using namespace std;

typedef double ld;
typedef long long LL;

const int chunk_size = 1<<16;


namespace io_impl
{
inline bool maybe_digit(char c)
{
    return c >= '0' && c <= '9';
}

struct io_s
{
private:
    FILE *fin;
    FILE *fout;

    bool negative;
    bool ok;
    char ch;

    inline char next_char()
    {
        static char buf[100000], *p1 = buf, *p2 = buf;
        return p1 == p2 && (p2 = (p1 = buf) + fread(buf, 1, 100000, fin), p1 == p2) ? EOF : *p1++;
    }

public:
    void init(FILE *_in, FILE *_out)
    {
        fin = _in;
        fout = _out;
        ch = next_char();
        ok = true;
    }

    template <typename T>
    bool run(T &_v)
    {
        _v = 0;
        while (!maybe_digit(ch) && ch != EOF)
            ch = next_char();
        if (ch == EOF)
            return ok = false;
        do
        {
            _v = (_v << 1) + (_v << 3) + ch - '0';
        } while (maybe_digit(ch = next_char()));
        return true;
    }

    template <typename T>
    bool rd(T &_v)
    {
        negative = false;
        _v = 0;
        while (!maybe_digit(ch) && ch != EOF)
        {
            negative = ch == '-';
            ch = next_char();
        }
        if (ch == EOF)
            return ok = false;
        do
        {
            _v = (_v * 10) + (ch - '0');
        } while (maybe_digit(ch = next_char()));
        static double _map[] = {1, 1e-1, 1e-2, 1e-3, 1e-4, 1e-5, 1e-6};
        if (ch == '.')
        {
            int tp = 0;
            while (maybe_digit(ch = next_char()))
            {
                _v = (_v * 10) + (ch - '0');
                ++tp;
            }
            _v *= _map[tp];
        }
        if (negative)
            _v = -_v;
        return true;
    }
    
};

} // namespace io_impl

using namespace io_impl;

io_s iokb;

namespace output {
    const int OutputBufferSize = 1 << 20;

    char buffer[OutputBufferSize];
    char *s = buffer;
    inline void flush() {
        fwrite(buffer, 1, s-buffer, stdout);
        s = buffer;
        fflush(stdout);
    }
    inline void print(const char ch) {
        // putchar(ch); return;
        if (s-buffer>OutputBufferSize-2) flush();
        *s++ = ch;
    }
    inline void print(char *str) {
        while (*str!=0) print(char(*str++));
    }
    inline void print(int x) {
        // printf("%d", x); return;
        char buf[25] = {0}, *p = buf;
        // if (x<0) print('-'), x=-x;
        // if (x == 0) print('0');
        while (x) *(++p) = x%10, x/=10;
        while (p != buf) print(char(*(p--)+'0'));
    }

    inline void print(LL x) {
        // printf("%d", x); return;
        char buf[25] = {0}, *p = buf;
        
        if (x == 0) print('0');
        while (x) *(++p) = x%10, x/=10;
        while (p != buf) print(char(*(p--)+'0'));
    }

    inline void print(ld v) {
        // printf("%.2f", x);
        // static int stk[70], tp;
        // tp = 0;
        if (v < 1e18) {
            if (fabs(v) < 0.005)
            {
                print('0');
                return;
            }
            else
            {
                LL x = (LL)floor(v * 100 + 0.5);
                if (x<0) print('-'), x=-x;
                // cerr << "x=" << x << endl; exit(0);
                print((LL)(x / 100));
                print('.');
                print((char)(x / 10 % 10 + '0'));
                print((char)(x % 10 + '0'));
            }
        } else {
            static char buf[30];
            sprintf(buf, "%.2lf", v);
            print(buf);
        }
        
    }
}



struct ios {
    
    inline ios & operator >> (int &x){
        iokb.run(x);
        return *this;
    }

   inline ios &operator>>(ld &x)
    {
        iokb.rd(x);
        return *this;
    }
} io;

inline void handleCudaError(hipError_t err, string name = "fuck") {
    if (err != hipSuccess) {
        cerr << name << endl;
        cerr << hipGetErrorString(err) << endl;
        exit(0);
    }
}

const int B = 8;

ld *d_a, *d_b, *d_c, *h_a, *h_b, *h_c;
int an, am, bn, bm;
int n, m;

void copyMatrix(ld *&src,  ld *&dst, int n, int m) {
    int size = sizeof(ld) * n * m;

    src = (ld*)malloc(size);

    for (int i=0; i<n; ++i)
    for (int j=0; j<m; ++j)
        io >> src[i * m + j];
    
    handleCudaError(hipMalloc(&dst, size), "hipMalloc in copyMatrix");
    handleCudaError(hipMemcpy(dst, src, size, hipMemcpyHostToDevice), "memcpy in copyMatrix");
}

void copyMatrixAsync(ld *&src, ld *&dst, int n, int m, hipStream_t &stream) {
    int size = sizeof(ld) * n * m;
    handleCudaError(hipMalloc(&dst, size), "hipMalloc in copyMatrix");
    handleCudaError(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream), "memcpyasync in copyMatrix");
}

template<typename T>
__global__ void matrixMult(T *d_a, T *d_b, T *d_c, int an, int bm, int am) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i = index / bm, j = index % bm;
    if (i >= an || j >= bm) return;
    register ld sum = 0;
    int basea = i * am;
   
    for (int k=0; k<am; ++k)
        sum += d_a[basea + k] * d_b[k * bm + j];

    d_c[i * bm + j] = sum;
    // int index = threadIdx.x;
    // if (index < an * bm)
    //     d_c[index] = 1; 
}


void simk(int grids, int block_size, ld *d_a, ld *d_b, ld *d_c, int an, int bm, int am) {
    for (int blockIdxx=0; blockIdxx<grids; ++blockIdxx) {
        for (int threadIdxx=0; threadIdxx<block_size; ++threadIdxx) {
            // printf("%d %d\n", blockIdxx, threadIdxx);
            int blockid = blockIdxx,
            threadid = threadIdxx; 
            int i = threadid / B, j = threadid % B, tbm = (bm + B - 1) / B, tam = (am + B - 1) / B;
            int rowInA = blockid / tam * B + i;
            int colInB = blockid % tbm * B + j;
            
            // if (i == 1 && j == 0) puts("FUCK");
            printf("blockid=%d, threadid=%d, i=%d, j=%d, rowInA=%d, colInB=%d, an=%d, bm=%d, block_size=%d, B=%d, am=%d\n", blockIdxx, threadIdxx, i, j, rowInA, colInB, an, bm, block_size, B, am);

            if (rowInA < an && j < am) printf("fill a[%d][%d]\n", i, j);
            if (i < am && colInB < bm) printf("fill b[%d][%d]\n", i, j);


            if (rowInA < an && colInB < bm) printf("fill c[%d][%d]\n", rowInA, colInB);
        }
    }
    // exit(0);
}



__global__ void matrixMult2(ld *d_a, ld *d_b, ld *d_c, int an, int bm, int am) {

    __shared__ ld a[B][B], b[B][B];

    

    int blockid = blockIdx.x,
    threadid = threadIdx.x;
    int i = threadid / B, j = threadid % B;
    int tbm = (bm + B - 1) / B;
    int rowInA = blockid / tbm * B + i;
    int colInB = blockid % tbm * B + j;

    ld sum = 0;

    for (int sub=0; sub<(am + B - 1) / B; ++sub) {
        int x = rowInA, y = sub * B + j;
        if (x < an && y < am)
            a[i][j] = d_a[x * am + y];
        else
            a[i][j] = 0;
        
        x = sub * B + i; y = colInB;
        if (x < am && y < bm)
            b[i][j] = d_b[x * bm + y];
        else
            b[i][j] = 0;

        __syncthreads();

        for (int k=0; k<B; ++k)
            sum += a[i][k] * b[k][j];

        __syncthreads();
    }
    if (rowInA < an && colInB < bm)
        d_c[(rowInA) * bm + colInB] = sum;
}

void outputMatrix(ld *a, int n, int m) {
    for (int i=0; i<n; ++i) {
        int base = i * m;
        output::print(a[base]);
        for (int j=1; j<m; ++j) {
            output::print(',');
            output::print(a[base + j]);
        }
        output::print('\n');
    }
}


void outputinterval(ld *c, int l, int r) {
    if (l == 0) {
        output::print(c[l++]);
    }
    for (register int i=l; i<r; ++i) {
        if (i % m == 0) output::print('\n');
        else output::print(',');
        output::print(c[i]);
    }
}
void outputMatrixAsync(ld *&a, ld *&d_a, int n, int m) {



    int st = 0, ed = n * m;
    // printf("st=%d ed=%d, a=%p\n", st, ed, a); 
    hipStream_t stream[2];
    int mask = 0;
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);
    int size;
    
    for (; st<ed; st+=size, mask^=1) {
        size = min(chunk_size, ed - st);
        // printf("st=%d st+size=%d, mask=%d\n", st, st+size, mask);
        // handleCudaError(cudaMemcpy(a + st, d_a + st, size * sizeof(ld), cudaMemcpyDeviceToHost));
        handleCudaError(hipMemcpyAsync(a + st, d_a + st, size * sizeof(ld), hipMemcpyDeviceToHost, stream[mask]));
        // exit(0);
        if (st - chunk_size >= 0) {
            // printf("%d %d\n",st-chunk_size, st);
            handleCudaError(hipStreamSynchronize(stream[mask^1]));
            outputinterval(a, st-chunk_size, st);
        }
    }
    st -= size;
    // sleep(1000);
    handleCudaError(hipStreamSynchronize(stream[0]), "sync stream0 last");
    handleCudaError(hipStreamSynchronize(stream[1]), "sync stream1 last");
    
    outputinterval(a, st, ed);
    output::print('\n');
}

void build(ld *&h, ld *&d, int n, int m, hipStream_t &s) {
    handleCudaError(hipHostAlloc(&h, sizeof(ld) * n * m, hipHostMallocDefault));

    for (int i=0; i<n; ++i) {
        for (int j=0; j<m; ++j) {
            io >> h[i * m + j];
        }
    }
    copyMatrixAsync(h, d, n, m, s);
}

int main()
{
    freopen("output.txt", "w", stdout);
    iokb.init(fopen("input.txt", "r"), fopen("output.txt", "w"));


    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cerr << prop.name << endl;

    // cudaStream_t mainstream;
    // cudaStreamCreate(&mainstream);

    // #endif
    io >> an >> am;
    // build(h_a, d_a, an, am, mainstream);
    copyMatrix(h_a, d_a, an, am);
    
    io >> bn >> bm; 
    // build(h_b, d_b, bn, bm, mainstream);
    copyMatrix(h_b, d_b, bn, bm);
    handleCudaError(hipMalloc(&d_c, sizeof(ld) * an * bm), "allocate for d_c");

    // handleCudaError(cudaStreamSynchronize(mainstream));


    int m = (an + B - 1) / B, n = (am + B - 1) / B, k = (bm + B - 1) / B;

    // simk(m * k, B * B, d_a, d_b, d_c, an, bm, am);

    fprintf(stderr, "stderr: m=%d, n=%d, k=%d\n", m, n, k);
    matrixMult2<<<m * k, B * B>>>(d_a, d_b, d_c, an, bm, am);

    handleCudaError(hipGetLastError(), "kernel error");
    fprintf(stderr, "stderr: running kernel completed\n");


    h_c = (ld*)malloc(sizeof(ld) * an * bm);
    // handleCudaError(cudaHostAlloc(&h_c, sizeof(ld) * an * bm,cudaHostAllocDefault), "hostalloc for c");
    handleCudaError(hipMemcpy(h_c, d_c, sizeof(ld) * an * bm, hipMemcpyDeviceToHost), "mem back");
    outputMatrix(h_c, an, bm);
    output::flush();
    
    return 0;
}



